#include "hip/hip_runtime.h"
#include "HedgehogConstraints.cuh"



template<class VFType, class ShiftType>
__global__ void HedgehogKernel_shrdmem(unsigned long int * constraints, const VFType *__restrict__ vectorField
	, unsigned __int64 inX, unsigned __int64 inY, unsigned __int64 inZ
	, ShiftType *in_shifts, double *in_nrm_shifts, unsigned long int inRadius, unsigned __int64 in_n_shifts, unsigned __int64 inconstriant_Nhs, double theta)
{
	extern __shared__ long int shared_mem[];
	__shared__ double * nrm_shifts;
	__shared__ ShiftType * shifts;
	double gamma = 90 - theta;
	unsigned __int64 n_voxels = inX*inY*inZ, n_shifts = in_n_shifts, X = inX, Y = inY, Z = inZ, XY = X*Y;
	unsigned __int64 r = inRadius, twoR1 = 2 * r + 1, twoR1sq = twoR1*twoR1, q, constriant_Nhs = inconstriant_Nhs;
	__int64 qx, qy, qz, px, py, pz;
	__int64 coutZ = inZ == 1 ? 0 : 1, nIdx, GIndx;
	double delta = 0;
	
	if (threadIdx.x == 0)
	{
		shifts = (ShiftType*) & shared_mem[0];
		memcpy(shifts, in_shifts, sizeof(long int)*n_shifts * 3);
		nrm_shifts = (double *)& shared_mem[n_shifts * 3];
		memcpy(nrm_shifts, in_nrm_shifts, sizeof(double)*n_shifts * 3);
	}
	__syncthreads();
	double vx, vy, vz;
	
	for (unsigned __int64 p = threadIdx.x + blockIdx.x*blockDim.x; p < n_voxels; p += blockDim.x*gridDim.x)
	{
		pz = p / XY;
		py = (p - pz*XY) / X;
		px = p - py*X - pz*XY;

		vx = vectorField[p * 3 + 0];
		vy = vectorField[p * 3 + 1];
		vz = vectorField[p * 3 + 2];
		for (unsigned long int s = 0; s < n_shifts; ++s)
		{
			delta = acos(nrm_shifts[s * 3 + 0] * vx + nrm_shifts[s * 3 + 1] * vy + nrm_shifts[s * 3 + 2] * vz) / HIP_PI_F*180;
			nIdx = shifts[s * 3 + 0] +r + (shifts[s * 3 + 1] + r)*twoR1 + (shifts[s * 3 + 2] + r)*twoR1sq*coutZ;
			if (delta <= gamma)
			{
				GIndx = p*constriant_Nhs + nIdx;
				atomicOr( (unsigned int *)(constraints + (GIndx >> 2)), 1 << ((GIndx % 4) * 8));  //polar cone
				qx = px - shifts[s * 3 + 0]; if (qx < 0 || qx >= X) continue; //might be able to ignore that condition to avoid diverging threads
				qy = py - shifts[s * 3 + 1]; if (qy < 0 || qy >= Y) continue; //might be able to ignore that condition to avoid diverging threads
				qz = pz - shifts[s * 3 + 2]; if (qz < 0 || qz >= Z) continue; //might be able to ignore that condition to avoid diverging threads
				q = qx + qy*X + qz*XY;
				GIndx =  q*constriant_Nhs + nIdx;
				atomicOr((unsigned int *) (constraints + (GIndx >> 2)), 1 << ((GIndx % 4) * 8));  //dual-cone of the-polar cone
			}
		}
	}
}
template<class VFType, class ShiftType>
__global__ void HedgehogKernel_gblm(unsigned long int * constraints, const VFType *__restrict__ vectorField
	, unsigned __int64 inX, unsigned __int64 inY, unsigned __int64 inZ
	, const ShiftType * __restrict__ shifts, const double *__restrict__ nrm_shifts, unsigned long int inRadius, unsigned __int64 in_n_shifts, unsigned __int64 inconstriant_Nhs, double theta)
{
	double gamma = 90 - theta;
	unsigned __int64 n_voxels = inX*inY*inZ, n_shifts = in_n_shifts, X = inX, Y = inY, Z = inZ, XY = X*Y;
	unsigned __int64 r = inRadius, twoR1 = 2 * r + 1, twoR1sq = twoR1*twoR1, q, constriant_Nhs = inconstriant_Nhs;
	__int64 qx, qy, qz, px, py, pz;
	__int64 coutZ = inZ == 1 ? 0 : 1, GIndx;
	double delta = 0 ;
	long int  nIdx;
	double vx, vy, vz;

	for (unsigned __int64 p = threadIdx.x + blockIdx.x*blockDim.x; p < n_voxels; p += blockDim.x*gridDim.x)
	{
		pz = p / XY;
		py = (p - pz*XY) / X;
		px = p - py*X - pz*XY;
		vx = vectorField[p * 3 + 0];
		vy = vectorField[p * 3 + 1];
		vz = vectorField[p * 3 + 2];
		for (unsigned long int s = 0; s < n_shifts; ++s)
		{
			delta = acos(nrm_shifts[s * 3 + 0] * vx + nrm_shifts[s * 3 + 1] * vy + nrm_shifts[s * 3 + 2] * vz) / HIP_PI_F * 180;
			nIdx = shifts[s * 3 + 0] + r + (shifts[s * 3 + 1] + r)*twoR1 + (shifts[s * 3 + 2] + r)*twoR1sq*coutZ;
			if (delta <= gamma)
			{
				GIndx =  p*constriant_Nhs + nIdx;
				atomicOr((unsigned int *)(constraints + (GIndx >> 2)), 1 << ((GIndx % 4) * 8));  //polar cone
				qx = px - shifts[s * 3 + 0]; if (qx < 0 || qx >= X) continue; //might be able to ignore that condition to avoid diverging threads
				qy = py - shifts[s * 3 + 1]; if (qy < 0 || qy >= Y) continue; //might be able to ignore that condition to avoid diverging threads
				qz = pz - shifts[s * 3 + 2]; if (qz < 0 || qz >= Z) continue; //might be able to ignore that condition to avoid diverging threads
				q = qx + qy*X + qz*XY;
				GIndx = q*constriant_Nhs + nIdx;
				atomicOr((unsigned int *)(constraints + (GIndx >> 2)), 1 << ((GIndx % 4) * 8));  //dual-cone of the-polar cone
			}
		}
	}
}
template<class VFType, class ShiftType>
Array2D<char> * getHedgehogConstraints(NDField<VFType> * vectorField, Array2D<ShiftType> * hhogshifts, unsigned long int hhog_radius, double theta)
{
	double mag;
	VFType * dev_vectorField = nullptr;
	unsigned long int * dev_constraints = nullptr;
	ShiftType * dev_hhogshifts = nullptr;
	double * dev_nrm_hhogshifts = nullptr;
	unsigned __int64 nVoxels = vectorField->X*vectorField->Y*vectorField->Z, nNhs = hhogshifts->Y;
	unsigned __int64 constriant_Nhs = pow2(2 * hhog_radius + 1);
	if (vectorField->Z > 1)
		constriant_Nhs *= 2 * hhog_radius + 1;
	Array2D<char> * constraintArray = new Array2D<char>(); constraintArray->allocate(constriant_Nhs, nVoxels);

	Array2D<double> nrm_hhogshifts; nrm_hhogshifts.allocate(hhogshifts->X, hhogshifts->Y);
	for (auto s = 0; s < hhogshifts->Y; ++s)
	{
		mag = sqrt(pow2(hhogshifts->data[3 * s + 0]) + pow2(hhogshifts->data[3 * s + 1]) + pow2(hhogshifts->data[3 * s + 2]));
		nrm_hhogshifts.data[3 * s + 0] = hhogshifts->data[3 * s + 0] / mag;
		nrm_hhogshifts.data[3 * s + 1] = hhogshifts->data[3 * s + 1] / mag;
		nrm_hhogshifts.data[3 * s + 2] = hhogshifts->data[3 * s + 2] / mag;
	}
	
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "Hedgehog: hipSetDevice failed!"); exit(EXIT_FAILURE); }

	cudaStatus = hipMalloc((void**)&dev_vectorField, sizeof(VFType)*vectorField->totalsize);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "Hedgehog: hipMalloc failed!"); exit(EXIT_FAILURE); }
	cudaStatus = hipMalloc((void**)&dev_hhogshifts, sizeof(ShiftType)*hhogshifts->totalsize);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "Hedgehog: hipMalloc failed!"); exit(EXIT_FAILURE); }
	cudaStatus = hipMalloc((void**)&dev_nrm_hhogshifts, sizeof(double)*nrm_hhogshifts.totalsize);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "Hedgehog: hipMalloc failed!"); exit(EXIT_FAILURE); }
	
	
	size_t dev_constraint_sizeinInts = nVoxels*constriant_Nhs/4;
	if ((nVoxels*constriant_Nhs) % 4 != 0)
		dev_constraint_sizeinInts++;
	cudaStatus = hipMalloc((void**)&dev_constraints, sizeof(unsigned long int)*dev_constraint_sizeinInts);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "Hedgehog: hipMalloc failed!"); exit(EXIT_FAILURE); }
	cudaStatus = hipMemset(dev_constraints, 0, sizeof(unsigned long int)*dev_constraint_sizeinInts);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "Hedgehog: hipMemset failed!"); exit(EXIT_FAILURE); }


	cudaStatus = hipMemcpyAsync(dev_vectorField, vectorField->field, sizeof(VFType)*vectorField->totalsize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "Hedgehog: hipMemcpy failed!"); exit(EXIT_FAILURE); }
	cudaStatus = hipMemcpyAsync(dev_hhogshifts, hhogshifts->data, sizeof(ShiftType)*hhogshifts->totalsize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "Hedgehog: hipMemcpy failed!"); exit(EXIT_FAILURE); }
	cudaStatus = hipMemcpyAsync(dev_nrm_hhogshifts, nrm_hhogshifts.data, sizeof(double)*nrm_hhogshifts.totalsize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "Hedgehog: hipMemcpy failed!"); exit(EXIT_FAILURE); }

	long int nBlocks = 32;
	long int nThreads = 128;
	size_t sharedmem_perblock = (sizeof(double)*hhogshifts->totalsize + sizeof(long int)*hhogshifts->totalsize);
	size_t total_sharedmem = nBlocks*sharedmem_perblock;
	if (total_sharedmem <= 46*1024 )
		HedgehogKernel_shrdmem << <nBlocks, nThreads, (unsigned long int)(sharedmem_perblock) >> >(dev_constraints, dev_vectorField, vectorField->X, vectorField->Y, vectorField->Z, dev_hhogshifts, dev_nrm_hhogshifts, hhog_radius, nNhs, constriant_Nhs, theta);
	else 
		HedgehogKernel_gblm << <nBlocks, nThreads >> >(dev_constraints, dev_vectorField, vectorField->X, vectorField->Y, vectorField->Z, dev_hhogshifts, dev_nrm_hhogshifts, hhog_radius, nNhs, constriant_Nhs, theta);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Hedgehog: Kernel failed error code %d after launching Kernel!\n", cudaStatus);
		std::cout << hipGetErrorString(cudaStatus) << std::endl;
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Hedgehog: hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
		std::cout << hipGetErrorString(cudaStatus) << std::endl;
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipMemcpy(constraintArray->data, dev_constraints, sizeof(char)*nVoxels * constriant_Nhs, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Hedgehog: output hipMemcpy failed!\n");
		std::cout << hipGetErrorString(cudaStatus)<<std::endl;
		exit(EXIT_FAILURE);
	}
	if (dev_vectorField)     { hipFree(dev_vectorField);      dev_vectorField = nullptr; }
	if (dev_hhogshifts)      { hipFree(dev_hhogshifts);       dev_hhogshifts = nullptr; }
	if (dev_nrm_hhogshifts)  { hipFree(dev_nrm_hhogshifts);   dev_nrm_hhogshifts = nullptr; }
	if (dev_constraints)     { hipFree(dev_constraints);      dev_constraints = nullptr; }

	return constraintArray;
}

template Array2D<char> * getHedgehogConstraints(NDField<float> *, Array2D<long int> *, unsigned long int, double);